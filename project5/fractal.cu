#include "hip/hip_runtime.h"
// Elliott Esponda & Andrew Wheeler
// (1.) Copied
#include <cstdlib>
#include <cmath>
#include <sys/time.h>
#include "cs43805351.h"
#include <hip/hip_runtime.h> // (2.) Done

static const double Delta = 0.004;
static const double xMid =  0.2389;
static const double yMid =  0.55267;
static const int ThreadsPerBlock = 512; // (3.) Done

// (4.) meet fractalKernel
static __global__ void fractalKernel(const int width, const int frames, unsigned char* pic)
{
  // compute frames
  const int pixels = frames * width * width;
	const int idx = threadIdx.x + blockIdx.x * blockDim.x; 
	if(idx < pixels) // (6.) Don't use excess
	{ // (5.) Bye loops, hello constants
		const int frame = idx / (width * width);
	  const int row = (idx / width) % width;
	  const int col = idx % width;
	  //for (int frame = 0; frame < frames; frame++) { 
	  const double delta = Delta * pow(0.98, frame);
	  const double xMin = xMid - delta;
		const double yMin = yMid - delta;
	  const double dw = 2.0 * delta / width;
	  //for (int row = 0; row < width; row++) {
	  const double cy = yMin + row * dw;
	  //for (int col = 0; col < width; col++) {
	  const double cx = xMin + col * dw;
	  double x = cx;
	  double y = cy;
	  int depth = 256;
	  double x2, y2;
	  do {
	    x2 = x * x;
	    y2 = y * y;
	    y = 2 * x * y + cy;
	    x = x2 - y2 + cx;
	    depth--;
	  } while ((depth > 0) && ((x2 + y2) < 5.0));
	  pic[frame * width * width + row * width + col] = (unsigned char)depth;
	}
}

static void CheckCuda()
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d: %s\n", e, hipGetErrorString(e));
    exit(-1);
  }
}


int main(int argc, char *argv[])
{
  printf("Fractal v1.7\n");

  // check command line
  if (argc != 3) {fprintf(stderr, "usage: %s frame_width num_frames\n", argv[0]); exit(-1);}
  const int width = atoi(argv[1]);
  if (width < 10) {fprintf(stderr, "error: frame_width must be at least 10\n"); exit(-1);}
  const int frames = atoi(argv[2]);
  if (frames < 1) {fprintf(stderr, "error: num_frames must be at least 1\n"); exit(-1);}
  printf("computing %d frames of %d by %d fractal\n", frames, width, width);
	
	// alloc space for device copy of pic (7.)
	const int N = frames * width * width;
  unsigned char * d_pic;
  const int size = N * sizeof(unsigned char); 
	hipMalloc((void **)&d_pic, size);

	// allocate picture array (host copies)
	unsigned char* pic = new unsigned char[N];

  // copy inputs to device
  if (hipSuccess != hipMemcpy(d_pic, pic, size, hipMemcpyHostToDevice)) {fprintf(stderr, "copying to device failed\n"); exit(-1);}
	
  // start time
  timeval start, end;
  gettimeofday(&start, NULL);

  // launch GPU kernel (8.) 
  fractalKernel<<<(N + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(width, frames, d_pic);
  hipDeviceSynchronize(); // (9.) Called

  // end time
  gettimeofday(&end, NULL);
  const double runtime = end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0;
  printf("compute time: %.3f s\n", runtime);	
	CheckCuda(); // (10.) CheckCuda

	// copy result back to host
  if (hipSuccess != hipMemcpy(pic, d_pic, size, hipMemcpyDeviceToHost)) {fprintf(stderr, "copying from device failed\n"); exit(-1);}
	
  // verify result by writing frames to BMP files
  if ((width <= 256) && (frames <= 100)) {
    for (int frame = 0; frame < frames; frame++) {
      char name[32];
      sprintf(name, "fractal%d.bmp", frame + 1000);
      writeBMP(width, width, &pic[frame * width * width], name);
    }
  }

  delete [] pic;
  hipFree(d_pic);
  return 0;
}

